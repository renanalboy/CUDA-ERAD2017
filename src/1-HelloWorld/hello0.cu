
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void hello()
{
	printf("Oi mundo! De: thread %d\n", threadIdx.x);
}

int main(void)
{
	int num_threads = 2;
	int num_blocks = 2;
	hello<<<num_blocks,num_threads>>>();
	hipDeviceSynchronize();

	return 0;
}
