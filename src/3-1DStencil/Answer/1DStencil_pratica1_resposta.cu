
#include <hip/hip_runtime.h>
#include <stdio.h>

#define RADIUS        3
#define BLOCK_SIZE    5
#define NUM_ELEMENTS  10

// CUDA API error checking macro
#define cudaCheck(error) \
  if (error != hipSuccess) { \
    printf("Fatal error: %s at %s:%d\n", \
      hipGetErrorString(error), \
      __FILE__, __LINE__); \
    exit(1); \
  }

__global__ void stencil_1d(int *in, int *out) 
{
    int gindex = threadIdx.x + (blockIdx.x * blockDim.x) + RADIUS;
    int local = 0;

    // Apply the stencil
    for (int offset = -RADIUS ; offset <= RADIUS ; offset++)
        local += in[gindex + offset];

    __syncthreads();
    out[gindex-RADIUS] = local;
    
}

int main()
{
  unsigned int i;
  int h_in[NUM_ELEMENTS + 2 * RADIUS], h_out[NUM_ELEMENTS];
  int *d_in, *d_out;

  // Initialize host data
  for( i = 0; i < (NUM_ELEMENTS + 2*RADIUS); ++i )
    h_in[i] = 1; // With a value of 1 and RADIUS of 3, all output values should be 7

  // Allocate space on the device
  cudaCheck( hipMalloc( &d_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int)) );
  cudaCheck( hipMalloc( &d_out, NUM_ELEMENTS * sizeof(int)) );

  // Copy input data to device
  cudaCheck( hipMemcpy( d_in, h_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int), hipMemcpyHostToDevice) );

  stencil_1d<<< (NUM_ELEMENTS + BLOCK_SIZE - 1)/BLOCK_SIZE, BLOCK_SIZE >>> (d_in, d_out);

  cudaCheck( hipMemcpy( h_out, d_out, NUM_ELEMENTS * sizeof(int), hipMemcpyDeviceToHost) );

  // Verify every out value is 7
  for( i = 0; i < NUM_ELEMENTS; ++i )
    if (h_out[i] != 7)
    {
      printf("Element h_out[%d] == %d != 7\n", i, h_out[i]);
      break;
    }

  if (i == NUM_ELEMENTS)
    printf("SUCCESS!\n");

  // Free out memory
  hipFree(d_in);
  hipFree(d_out);

  return 0;
}

