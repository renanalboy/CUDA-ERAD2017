
#include <hip/hip_runtime.h>
#include <stdio.h>

#define RADIUS        3
#define BLOCK_SIZE    256
#define NUM_ELEMENTS  (4096*2)

// CUDA API error checking macro
#define cudaCheck(error) \
  if (error != hipSuccess) { \
    printf("Fatal error: %s at %s:%d\n", \
      hipGetErrorString(error), \
      __FILE__, __LINE__); \
    exit(1); \
  }

__global__ void stencil_1d(int *in, int *out) 
{
    /* COMPLETE COM A ALOCAÇÃO DE MEMÓRIA COMPARTILHADA*/

    int gindex = threadIdx.x + (blockIdx.x * blockDim.x) + RADIUS; //índice global
    int lindex = threadIdx.x + RADIUS; //índice local (a ser usado com o vetor alocado com sh mem)

    /*COMPLETE COM O CÓDIGO PARA COPIAR CADA ELEMENTO GLOBAL PARA A MEMÓRIA COMPARTILHADA LOCAL
    OBS: não esquecer da margem (halo)*/


    /*COMPLETE COM O CÓDIGO QUE CALCULA LOCALMENTE O STENCIL E COPIA O RESULTADO LOCAL PARA A MEMÓRIA GLOBAL*/

}

int main()
{
  unsigned int i;
  int h_in[NUM_ELEMENTS + 2 * RADIUS], h_out[NUM_ELEMENTS];
  int *d_in, *d_out;

  // Initialize host data
  for( i = 0; i < (NUM_ELEMENTS + 2*RADIUS); ++i )
    h_in[i] = 1; // With a value of 1 and RADIUS of 3, all output values should be 7

  // Allocate space on the device
  cudaCheck( hipMalloc( &d_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int)) );
  cudaCheck( hipMalloc( &d_out, NUM_ELEMENTS * sizeof(int)) );

  // Copy input data to device
  cudaCheck( hipMemcpy( d_in, h_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int), hipMemcpyHostToDevice) );

  stencil_1d<<< (NUM_ELEMENTS + BLOCK_SIZE - 1)/BLOCK_SIZE, BLOCK_SIZE >>> (d_in, d_out);

  cudaCheck( hipMemcpy( h_out, d_out, NUM_ELEMENTS * sizeof(int), hipMemcpyDeviceToHost) );

  // Verify every out value is 7
  for( i = 0; i < NUM_ELEMENTS; ++i )
    if (h_out[i] != 7)
    {
      printf("Element h_out[%d] == %d != 7\n", i, h_out[i]);
      break;
    }

  if (i == NUM_ELEMENTS)
    printf("SUCCESS!\n");

  // Free out memory
  hipFree(d_in);
  hipFree(d_out);

  return 0;
}

