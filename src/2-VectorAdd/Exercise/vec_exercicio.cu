#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void add(int *a, int *b, int *c)
{
    /* insira codigo para calcular o indice usando blockIdx.x, blockDim.x, threadIdx.x */
	int index = /* RESPONDA */
	c[index] = a[index] + b[index];
}

/* EXPERIMENTE com N */
/* o quao grande ele pode ser? */
#define N (2048*2048)
#define THREADS_PER_BLOCK 512

int main()
{
  int *a, *b, *c;
	int *d_a, *d_b, *d_c;

	int size = N * sizeof( int );

	/* aloca espaco no device para copias de a, b, c */
	hipMalloc( (void **) &d_a, size );
	hipMalloc( (void **) &d_b, size );
	hipMalloc( (void **) &d_c, size );

	/* aloca espaco no host para copias de a, b, c e inicializa valores */

	a = (int *)malloc( size );
	b = (int *)malloc( size );
	c = (int *)malloc( size );

	for( int i = 0; i < N; i++ )
	{
		a[i] = b[i] = i;
		c[i] = 0;
	}

	/* copia dados para o device */
	/* COMPLETE com os parametros necessários para copiar dados para o device */
	hipMemcpy( /* COMPLETE */ );
	hipMemcpy( /* COMPLETE */ );

	/* lance o kernel na GPU */
	/* COMPLETE com os parametros para lançar o kernel propriamente usando blocos e threads */
	add<<< /* COMPLETE */, /* COMPLETE */ >>>( d_a, d_b, d_c );

	/* copie os resultados de volta ao host */
	/* COMPLETE com parametros necessarios para copiar os dados de volta ao host */
	hipMemcpy( /* COMPLETE */ );


	printf( "c[0] = %d\n", c[0] );
	printf( "c[%d] = %d\n",N-1, c[N-1] );

	/* clean up */

	free(a);
	free(b);
	free(c);
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
} /* end main */
