
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c)
{
    /* insert code to calculate the index properly using blockIdx.x, blockDim.x, threadIdx.x */
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	c[index] = a[index] + b[index];
}

/* experiment with N */
/* how large can it be? */
#define N (2048*2048)
#define THREADS_PER_BLOCK 512

int main()
{
	int *d_a, *d_b, *d_c;
	int size = N * sizeof( int );

	/* allocate space for device copies of a, b, c */

	hipMallocManaged(&d_a, size );
	hipMallocManaged(&d_b, size );
	hipMallocManaged(&d_c, size );

	for( int i = 0; i < N; i++ )
	{
		d_a[i] = d_b[i] = i;
		d_c[i] = 0;
	}

	/* launch the kernel on the GPU */
	/* insert the launch parameters to launch the kernel properly using blocks and threads */ 
	add<<< (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( d_a, d_b, d_c );

        hipDeviceSynchronize();

	printf( "d_c[0] = %d\n", d_c[0] );
	printf( "d_c[%d] = %d\n",N-1, d_c[N-1] );

	/* clean up */

	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
} /* end main */
