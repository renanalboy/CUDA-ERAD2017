
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
    *c = *a + *b;
}


int main(void) {
    int *d_a, *d_b, *d_c; // device copies of a, b, c
    int size = sizeof(int);

    // Allocate space for device copies of a, b, c
    hipMallocManaged(&d_a, size);
    hipMallocManaged (&d_b, size);
    hipMallocManaged (&d_c, size);

    // Setup input values
    *d_a = 2;
    *d_b = 7;
    *d_c = 0;

    // Launch add() kernel on GPU
    add<<<1,1>>>(d_a, d_b, d_c);

    hipDeviceSynchronize();

    printf("d_c = %d\n", *d_c);

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
